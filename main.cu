// -*- mode: C -*-

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "d_main.h"
#include "kernals.h"
#include "mt19937ar.h"


int main () {

    unsigned long rngseed = 2894203475;  // RNG seed (fixed for development/testing)
    //unsigned long rngseed = (long)time(NULL);
    // Initialise host RNG
    init_genrand(rngseed);

    // Get GPU info
    hipDeviceProp_t prop;
    int cudaDevice, \
        devGlobalMem, devSharedMemPerBlock, 
        devThreadsPerblock, devMultiProc,
        devConcurrentKernals;
    int maxThreads[3], maxGrid[3];
    hipGetDevice(&cudaDevice);
    printf("%c", (char)(cudaDevice));
    hipSetDevice(cudaDevice) ;
    // Really we should query property by property as this has some excess overhead,
    // while we dont know what properties we need this is preferable.
    hipGetDeviceProperties(&prop, cudaDevice);
    int devConcurrentKernels = prop.concurrentKernels;
    if (devConcurrentKernels == 0) {
        printf("Error, this code requires concurrant kernal launches CC>5");
        exit(1);
    }

    // Get Threads and blocks
    devGlobalMem = prop.totalGlobalMem;
    devSharedMemPerBlock = prop.sharedMemPerBlock;
    devThreadsPerblock = prop.maxThreadsPerBlock;
    devMultiProc = prop.multiProcessorCount;
       
    int N_bl = 5; //we are going to span 5 blocks
    int N_th = 5; //with 5 threads
    int N_child = 5; // which all launch 5 children


    // Initilise RNG on GPU
    hiprandState *d_state;
    hipMalloc((void **)&d_state, N_bl*N_th*sizeof(hiprandState));
    unsigned long long gpuseed = (unsigned long long)rngseed;

    // create global memory array for child output
    int *host_child_out;
    hipHostAlloc((void**)&host_child_out, N_child*N_th*N_bl*sizeof(int), hipHostMallocDefault);

    // Create varable to instuct dmain on how to launch children
    int *dev_N_child;
    hipMalloc( (void**)&dev_N_child, sizeof(N_child));


    // dynamically sized arrays
    int dev_child_out[N_bl];
    hipStream_t streams[N_bl];

    for (int i=0;i<N_bl;i++){
        hipStreamCreate(&streams[i]);
        hipMalloc((void**)dev_child_out[i], N_child*N_th*sizeof(int));
    }
    for (int i=0;i<N_bl;i++){
        // init the RNG
        init_gpurand<<<1,N_th,0,streams[i]>>>(gpuseed, N_bl, d_state);
    }
    for (int i=0;i<N_bl;i++){
        // Launch d_main
        hipMemcpyAsync(dev_N_child, N_child, hipMemcpyDeviceToHost,  streams[i]);
    }
    for (int i=0;i<N_bl;i++){
        d_main<<<1, N_th,0,streams[i]>>>(N_child, d_state, dev_child_out[i]);
    }

    for (int i=0;i<N_bl;i++){
        cudaMemCpyAsync(host_child_out+i*N_child*N_th, dev_child_out[i], N_child*N_th*sizeof(int), streams[i]);
    }

    //Synchronise

    for (int i=0;i<N_bl;i++){
        hipFree(dev_child_out[i]);
        cudaStreamDestory(streams[i]);
    }


    printf(host_child_out);
    // Free the memory
    hipHostFree(host_child_out);
    hipFree(dev_N_child);

    // Output
    printf("Done!");

    return 0;

}